
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <stdlib.h>
#include <stdint.h>
#include <sys/time.h>
#include <cmath>
#include <math.h>
#include <sys/stat.h>
#include <cstdio>

/**
    srun -A bchn-delta-gpu --time=00:20:00 --nodes=1 --tasks-per-node=16 --partition=gpuA100x4,gpuA40x4 --gpus=1 --mem=16g --pty /bin/bash
*/

static const uint8_t sbox[256] = {
    // 0     1    2      3     4    5     6     7      8    9     A      B    C     D     E     F
    0x63, 0x7c, 0x77, 0x7b, 0xf2, 0x6b, 0x6f, 0xc5, 0x30, 0x01, 0x67, 0x2b, 0xfe, 0xd7, 0xab, 0x76,
    0xca, 0x82, 0xc9, 0x7d, 0xfa, 0x59, 0x47, 0xf0, 0xad, 0xd4, 0xa2, 0xaf, 0x9c, 0xa4, 0x72, 0xc0,
    0xb7, 0xfd, 0x93, 0x26, 0x36, 0x3f, 0xf7, 0xcc, 0x34, 0xa5, 0xe5, 0xf1, 0x71, 0xd8, 0x31, 0x15,
    0x04, 0xc7, 0x23, 0xc3, 0x18, 0x96, 0x05, 0x9a, 0x07, 0x12, 0x80, 0xe2, 0xeb, 0x27, 0xb2, 0x75,
    0x09, 0x83, 0x2c, 0x1a, 0x1b, 0x6e, 0x5a, 0xa0, 0x52, 0x3b, 0xd6, 0xb3, 0x29, 0xe3, 0x2f, 0x84,
    0x53, 0xd1, 0x00, 0xed, 0x20, 0xfc, 0xb1, 0x5b, 0x6a, 0xcb, 0xbe, 0x39, 0x4a, 0x4c, 0x58, 0xcf,
    0xd0, 0xef, 0xaa, 0xfb, 0x43, 0x4d, 0x33, 0x85, 0x45, 0xf9, 0x02, 0x7f, 0x50, 0x3c, 0x9f, 0xa8,
    0x51, 0xa3, 0x40, 0x8f, 0x92, 0x9d, 0x38, 0xf5, 0xbc, 0xb6, 0xda, 0x21, 0x10, 0xff, 0xf3, 0xd2,
    0xcd, 0x0c, 0x13, 0xec, 0x5f, 0x97, 0x44, 0x17, 0xc4, 0xa7, 0x7e, 0x3d, 0x64, 0x5d, 0x19, 0x73,
    0x60, 0x81, 0x4f, 0xdc, 0x22, 0x2a, 0x90, 0x88, 0x46, 0xee, 0xb8, 0x14, 0xde, 0x5e, 0x0b, 0xdb,
    0xe0, 0x32, 0x3a, 0x0a, 0x49, 0x06, 0x24, 0x5c, 0xc2, 0xd3, 0xac, 0x62, 0x91, 0x95, 0xe4, 0x79,
    0xe7, 0xc8, 0x37, 0x6d, 0x8d, 0xd5, 0x4e, 0xa9, 0x6c, 0x56, 0xf4, 0xea, 0x65, 0x7a, 0xae, 0x08,
    0xba, 0x78, 0x25, 0x2e, 0x1c, 0xa6, 0xb4, 0xc6, 0xe8, 0xdd, 0x74, 0x1f, 0x4b, 0xbd, 0x8b, 0x8a,
    0x70, 0x3e, 0xb5, 0x66, 0x48, 0x03, 0xf6, 0x0e, 0x61, 0x35, 0x57, 0xb9, 0x86, 0xc1, 0x1d, 0x9e,
    0xe1, 0xf8, 0x98, 0x11, 0x69, 0xd9, 0x8e, 0x94, 0x9b, 0x1e, 0x87, 0xe9, 0xce, 0x55, 0x28, 0xdf,
    0x8c, 0xa1, 0x89, 0x0d, 0xbf, 0xe6, 0x42, 0x68, 0x41, 0x99, 0x2d, 0x0f, 0xb0, 0x54, 0xbb, 0x16};

__constant__ uint8_t constant_sbox[256];
__constant__ uint8_t constantRoundKey[176];

/******************************************************************************************************* */
/* Helper Functions*/
/* START */

#define BLOCK_SIZE 16
#define Nk 4  // The number of 32 bit words in a key.
#define Nr 10 // The number of rounds in AES Cipher.
#define getSBoxValue(num) (sbox[(num)])
static const uint8_t Rcon[11] = {
    0x8d, 0x01, 0x02, 0x04, 0x08, 0x10, 0x20, 0x40, 0x80, 0x1b, 0x36};

#define CHECK(call)                                                                  \
    {                                                                                \
        const hipError_t cuda_ret = call;                                           \
        if (cuda_ret != hipSuccess)                                                 \
        {                                                                            \
            printf("Error: %s:%d, ", __FILE__, __LINE__);                            \
            printf("code: %d, reason:%s\n", cuda_ret, hipGetErrorString(cuda_ret)); \
            exit(-1);                                                                \
        }                                                                            \
    }

double myCPUTimer()
{
    struct timeval tp;
    gettimeofday(&tp, NULL);
    return ((double)tp.tv_sec + (double)tp.tv_usec / 1.0e6);
}

void padData(uint8_t *data, size_t file_size, size_t padded_size)
{
    uint8_t padding_value = padded_size - file_size;
    memset(data + file_size, padding_value, padding_value);
}

// This function produces Nb(Nr+1) round keys. The round keys are used in each round to decrypt the states.
static void keyExpansion(uint8_t *roundKey, const uint8_t *key)
{
    unsigned i, j, k;
    uint8_t tempa[4]; // Used for the column/row operations

    // The first round key is the key itself.
    for (i = 0; i < Nk; ++i)
    {
        roundKey[(i * 4) + 0] = key[(i * 4) + 0];
        roundKey[(i * 4) + 1] = key[(i * 4) + 1];
        roundKey[(i * 4) + 2] = key[(i * 4) + 2];
        roundKey[(i * 4) + 3] = key[(i * 4) + 3];
    }

    // All other round keys are found from the previous round keys.
    for (i = Nk; i < Nk * (Nr + 1); ++i)
    {

        k = (i - 1) * 4;
        tempa[0] = roundKey[k + 0];
        tempa[1] = roundKey[k + 1];
        tempa[2] = roundKey[k + 2];
        tempa[3] = roundKey[k + 3];

        if (i % Nk == 0)
        {
            // Rotate word
            const uint8_t u8tmp = tempa[0];
            tempa[0] = tempa[1];
            tempa[1] = tempa[2];
            tempa[2] = tempa[3];
            tempa[3] = u8tmp;

            // substitue bytes in word
            tempa[0] = getSBoxValue(tempa[0]);
            tempa[1] = getSBoxValue(tempa[1]);
            tempa[2] = getSBoxValue(tempa[2]);
            tempa[3] = getSBoxValue(tempa[3]);

            tempa[0] = tempa[0] ^ Rcon[i / Nk];
        }
        j = i * 4;
        k = (i - Nk) * 4;
        roundKey[j + 0] = roundKey[k + 0] ^ tempa[0];
        roundKey[j + 1] = roundKey[k + 1] ^ tempa[1];
        roundKey[j + 2] = roundKey[k + 2] ^ tempa[2];
        roundKey[j + 3] = roundKey[k + 3] ^ tempa[3];
    }
}

/* END */
/* Helper Functions*/
/******************************************************************************************************* */

/******************************************************************************************************* */
/* AES Functions*/
/* START */

__device__ void subBytes(uint8_t *state)
{

    for (int i = 0; i < 16; i++)
    {
        uint8_t byte = state[i];
        uint8_t first4Bits = (byte & 0xF0) >> 4;
        uint8_t last4Bits = byte & 0x0F;
        int sbox_index = (first4Bits * 16) + last4Bits;

        state[i] = constant_sbox[sbox_index];
    }
}

__device__ void shiftRows(uint8_t *state)
{
    uint8_t temp;

    temp = state[1];
    for (int i = 1; i <= 9; i += 4)
        state[i] = state[i + 4];
    state[13] = temp;

    temp = state[2];
    state[2] = state[10];
    state[10] = temp;
    temp = state[6];
    state[6] = state[14];
    state[14] = temp;

    temp = state[15];
    for (int i = 15; i >= 7; i -= 4)
        state[i] = state[i - 4];
    state[3] = temp;
}

__device__ __forceinline__ uint8_t xtime(uint8_t x)
{
    return (x << 1) ^ ((-(x >> 7)) & 0x1B);
}

__device__ void mixColumns(uint8_t *state)
{
    uint8_t temp[16];

#pragma unroll
    for (int col = 0; col < 4; col++)
    {
        int i = col * 4;

        uint8_t s0 = state[i];
        uint8_t s1 = state[i + 1];
        uint8_t s2 = state[i + 2];
        uint8_t s3 = state[i + 3];

        uint8_t xt0 = xtime(s0);
        uint8_t xt1 = xtime(s1);
        uint8_t xt2 = xtime(s2);
        uint8_t xt3 = xtime(s3);

        // MixColumns matrix multiplication in GF(2^8)
        temp[i + 0] = xt0 ^ (xt1 ^ s1) ^ s2 ^ s3; // 2*s0 + 3*s1 + s2   + s3
        temp[i + 1] = s0 ^ xt1 ^ (xt2 ^ s2) ^ s3; // s0   + 2*s1 + 3*s2 + s3
        temp[i + 2] = s0 ^ s1 ^ xt2 ^ (xt3 ^ s3); // s0   + s1   + 2*s2 + 3*s3
        temp[i + 3] = (xt0 ^ s0) ^ s1 ^ s2 ^ xt3; // 3*s0 + s1   + s2   + 2*s3
    }

#pragma unroll
    for (int i = 0; i < 16; i++)
    {
        state[i] = temp[i];
    }
}

__device__ void addRoundKey(uint8_t *state, int round)
{
    for (int i = 0; i < 16; i++)
    {
        state[i] ^= constantRoundKey[round * 16 + i];
    }
}

__global__ void encryptAes(uint8_t *in, uint8_t *out, unsigned int n)
{
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    int offset = idx * 16; // Each thread processes 16 bytes

    if (offset >= n)
        return;

    addRoundKey(in + offset, 0);

    for (int i = 1; i <= 9; i++)
    {
        subBytes(in + offset);
        shiftRows(in + offset);
        mixColumns(in + offset);
        addRoundKey(in + offset, i);
    }

    subBytes(in + offset);
    shiftRows(in + offset);
    addRoundKey(in + offset, 10);

    // Copy 16 bytes from input to output
    for (int i = 0; i < 16; i++)
    {
        if (offset + i < n)
        {
            out[offset + i] = in[offset + i];
        }
    }
}

/* AES Functions*/
/* END */
/******************************************************************************************************* */

int main(int argc, char *argv[])
{
    if (argc != 3)
    {
        printf("Usage: %s <input file> <output file>\n", argv[0]);
        return 1;
    }

    struct stat file_stat;
    if (stat(argv[1], &file_stat) != 0)
    {
        perror("stat");
        return 1;
    }
    size_t file_size = file_stat.st_size;
    size_t padded_size = (file_size % BLOCK_SIZE == 0) ? file_size : ((file_size / BLOCK_SIZE + 1) * BLOCK_SIZE);

    char *file_name = argv[1];
    printf("\nEncrypting file: \"%s\"\n", file_name);

    printf("\nFile Size: %d bytes\n", file_size);
    printf("\nPadded file Size: %d \n", padded_size);

    uint8_t *buffer = (uint8_t *)malloc(padded_size);
    if (!buffer)
    {
        perror("Memory allocation failed");
        return 1;
    }

    FILE *input_file = fopen(argv[1], "rb");
    if (!input_file)
    {
        perror("Failed to open input file");
        free(buffer);
        return 1;
    }
    fread(buffer, 1, file_size, input_file);
    fclose(input_file);

    padData(buffer, file_size, padded_size);

    // for (int i = 0; i < padded_size; i++) {
    //     for (int bit = 7; bit >= 0; bit--) {
    //         printf("%d", (buffer[i] >> bit) & 1);  // Extract and print each bit
    //     }
    //     printf(" ");  // Separate bytes with a space
    // }
    // printf("\n");

    // AES key
    uint8_t key[BLOCK_SIZE] = {
        0x2b, 0x7e, 0x15, 0x16, 0x28, 0xae, 0xd2, 0xa6,
        0xab, 0xf7, 0x15, 0x88, 0x09, 0xcf, 0x4f, 0x3c};

    uint8_t roundKey[176];
    keyExpansion(roundKey, key);

    printf("Key: ");
    for (int i = 0; i < 176; i++)
    {
        printf("%02X ", roundKey[i]);
    }
    printf("\n");

    printf("Encrypting...\n");
    uint8_t *inBuff, *outBuff;
    hipMalloc((void **)&inBuff, sizeof(uint8_t) * padded_size);
    hipMalloc((void **)&outBuff, sizeof(uint8_t) * padded_size);

    hipMemcpy(inBuff, buffer, sizeof(uint8_t) * padded_size, hipMemcpyHostToDevice);

    hipMemcpyToSymbol(HIP_SYMBOL(constant_sbox), sbox, 256 * sizeof(uint8_t));
    hipMemcpyToSymbol(HIP_SYMBOL(constantRoundKey), roundKey, 176 * sizeof(uint8_t));

    int numThreads = padded_size / 16;
    dim3 blockDim(16);
    dim3 gridDim((numThreads + 16 - 1) / 16);

    encryptAes<<<gridDim, blockDim>>>(inBuff, outBuff, padded_size);
    CHECK(hipDeviceSynchronize());

    uint8_t *outFileBuff = (uint8_t *)calloc(padded_size, sizeof(uint8_t));
    hipMemcpy(outFileBuff, outBuff, sizeof(uint8_t) * padded_size, hipMemcpyDeviceToHost);

    // Open the output file
    FILE *output_file = fopen(argv[2], "wb");
    if (!output_file)
    {
        perror("Failed to open output file");
        free(outFileBuff);
        hipFree(inBuff);
        hipFree(outBuff);
        free(buffer);
        return 1;
    }

    // Write to the output file
    fwrite(outFileBuff, sizeof(uint8_t), padded_size, output_file);
    fclose(output_file);

    printf("Encryption complete. Output written to \"%s\"\n", argv[2]);

    // printf("IN  OUT \n");
    // for(int i = 0; i < 32; i++){
    //     printf("%02X  %02X\n",buffer[i], outFileBuff[i]);
    // }

    hipFree(inBuff);
    hipFree(outBuff);
    free(outFileBuff);
    free(buffer);
    return 0;
}
